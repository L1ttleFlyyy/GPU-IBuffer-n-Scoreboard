
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct
{
    int width;
    int height;
    float *elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    hipSetDevice(0);
    hipDeviceSynchronize();
    size_t available, total;
    hipMemGetInfo(&available, &total);
    // printf("Mem total: %ld Bytes\nMem available: %ld Bytes\n", available, total);
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    // printf("size of A: %ld\n", size);
    hipMalloc(&d_A.elements, size);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation A %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation B %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    error = hipGetLastError();
    hipMalloc(&d_C.elements, size);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: calculation error %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: copying C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int myrand()
{
    return rand() / (RAND_MAX / 10);
}

int main()
{ // A x B
    srand(0);
    Matrix A, B, C;
    A.height = 1 * BLOCK_SIZE;
    A.width = 1 * BLOCK_SIZE; // hB = wA
    B.height = A.width;
    B.width = 1 * BLOCK_SIZE;
    C.height = A.height; // hC = hA
    C.width = B.width;   // wC = wB
    A.elements = (float *)malloc(A.height * A.width * sizeof(float));
    B.elements = (float *)malloc(B.height * B.width * sizeof(float));
    C.elements = (float *)malloc(C.height * C.width * sizeof(float));
    printf("Content of A: \n");
    for (int i = 0; i < A.height; i++)
    {
        for (int j = 0; j < A.width; j++)
        {
            A.elements[i * A.height + j] = myrand();
            printf("%2d", (int)A.elements[i * A.height + j]);
        }
        printf("\n");
    }

    printf("\n\nContent of B: \n");
    for (int i = 0; i < B.height; i++)
    {
        for (int j = 0; j < B.width; j++)
        {
            B.elements[i * B.height + j] = myrand();
            printf("%2d", (int)B.elements[i * B.height + j]);
        }
        printf("\n");
    }

    MatMul(A, B, C);

    printf("\n\nContent of C: \n");
    for (int i = 0; i < C.height; i++)
    {
        for (int j = 0; j < C.width; j++)
        {
            printf("%4d", (int)C.elements[i * C.height + j]);
        }
        printf("\n");
    }

    return 0;
}
