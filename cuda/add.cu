
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct
{
    int width;
    int height;
    float *elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatAddKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatAdd(const Matrix A, const Matrix B, Matrix C)
{
    hipSetDevice(0);
    hipDeviceSynchronize();
    size_t available, total;
    hipMemGetInfo(&available, &total);
    printf("Mem total: %ld Bytes\nMem available: %ld Bytes\n", available, total);
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    printf("size of A: %ld\n", size);
    hipMalloc(&d_A.elements, size);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation A %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    hipMalloc(&d_B.elements, size);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation B %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    error = hipGetLastError();
    hipMalloc(&d_C.elements, size);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: calculation error %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: copying C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatAddKernel(Matrix A, Matrix B, Matrix C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int id = row * A.width + col;
    C.elements[id] = A.elements[id] + B.elements[id];
}

int myrand()
{
    return rand() / (RAND_MAX / 10);
}

int main()
{ // A x B
    srand(0);
    Matrix A, B, C;
    A.height = B.height = C.height = 2 * BLOCK_SIZE;
    A.width = B.width = C.width = 1 * BLOCK_SIZE;
    A.elements = (float *)malloc(A.height * A.width * sizeof(float));
    B.elements = (float *)malloc(B.height * B.width * sizeof(float));
    C.elements = (float *)malloc(C.height * C.width * sizeof(float));
    printf("Content of A: \n");
    for (int i = 0; i < A.height; i++)
    {
        for (int j = 0; j < A.width; j++)
        {
            A.elements[i * A.height + j] = myrand();
            printf("%2d", (int)A.elements[i * A.height + j]);
        }
        printf("\n");
    }

    printf("\n\nContent of B: \n");
    for (int i = 0; i < B.height; i++)
    {
        for (int j = 0; j < B.width; j++)
        {
            B.elements[i * B.height + j] = myrand();
            printf("%2d", (int)B.elements[i * B.height + j]);
        }
        printf("\n");
    }

    MatAdd(A, B, C);

    printf("\n\nContent of C: \n");
    for (int i = 0; i < C.height; i++)
    {
        for (int j = 0; j < C.width; j++)
        {
            printf("%3d", (int)C.elements[i * C.height + j]);
        }
        printf("\n");
    }

    return 0;
}
