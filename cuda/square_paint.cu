
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct
{
    int width;
    int height;
    float *elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void PaintKernel(Matrix);


void Paint(Matrix C)
{
    hipSetDevice(0);
    hipDeviceSynchronize();
    size_t available, total;
    hipMemGetInfo(&available, &total);
    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size_t size = C.width * C.height * sizeof(float);
    hipError_t error = hipGetLastError();
    hipMalloc(&d_C.elements, size);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(C.width / dimBlock.x, C.height / dimBlock.y);
    PaintKernel<<<dimGrid, dimBlock>>>(d_C);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: calculation error %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // write C to device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: copying C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Free device memory
    hipFree(d_C.elements);
}


__global__ void PaintKernel(Matrix C)
{
    // Each thread computes one element of C
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col + row > BLOCK_SIZE/2)
        if (col + row < BLOCK_SIZE/2*3) 
            if (row - col < BLOCK_SIZE/2)
                if (row - col > -BLOCK_SIZE/2)
                        Cvalue = 1;
    C.elements[row * C.width + col] = Cvalue;
}


int main()
{
    Matrix C;
    C.height = 1 * BLOCK_SIZE; // hC = hA
    C.width = 1 * BLOCK_SIZE;   // wC = wB
    size_t nBytes = C.height * C.width * sizeof(float);
    C.elements = (float *)malloc(nBytes);
    memset(C.elements, 0, nBytes);

    Paint(C);
    printf("Num: \n");
    for (int i = 0; i < C.height; i++)
    {
        for (int j = 0; j < C.width; j++)
        {
            printf("%2d", (int)C.elements[i * C.height + j]);
        }
        printf("\n");
    }

    printf("\n\nPainting: \n");
    for (int i = 0; i < C.height; i++)
    {
        for (int j = 0; j < C.width; j++)
        {
            if(C.elements[i * C.height + j]) 
                printf(" -");
            else
                printf(" 0");
        }
        printf("\n");
    }

    return 0;
}
