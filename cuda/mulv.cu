
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct
{
    int width;
    int height;
    float *elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    hipSetDevice(0);
    hipDeviceSynchronize();
    size_t available, total;
    hipMemGetInfo(&available, &total);
    printf("Mem total: %ld Bytes\nMem available: %ld Bytes\n", available, total);
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    size_t size = A.width * sizeof(float);
    printf("size of A: %ld\n", size);
    hipMalloc(&d_A.elements, size);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation A %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    hipMalloc(&d_B.elements, size);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation B %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    error = hipGetLastError();
    hipMalloc(&d_C.elements, size);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: allocation C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(B.width / BLOCK_SIZE, 1);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: calculation error %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "ERROR: copying C %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    C.elements[id] = A.elements[id] * B.elements[id];
}

int myrand()
{
    return rand() / (RAND_MAX / 10);
}

int main()
{ // A x B
    srand(0);
    Matrix A, B, C;
    A.height = B.height = C.height = 1;
    A.width = B.width = C.width = 1 * BLOCK_SIZE;
    A.elements = (float *)malloc(A.width * sizeof(float));
    B.elements = (float *)malloc(B.width * sizeof(float));
    C.elements = (float *)malloc(C.width * sizeof(float));
    printf("Content of A: \n");
    for (int j = 0; j < A.width; j++)
    {
        A.elements[j] = myrand();
        printf("%2d", (int)A.elements[j]);
    }
    printf("\n");

    printf("\n\nContent of B: \n");
    for (int j = 0; j < B.width; j++)
    {
        B.elements[j] = myrand();
        printf("%2d", (int)B.elements[j]);
    }
    printf("\n");

    MatMul(A, B, C);

    printf("\n\nContent of C: \n");
    for (int j = 0; j < C.width; j++)
    {
        printf("%3d", (int)C.elements[j]);
    }
    printf("\n");
    return 0;
}
